#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdint.h>

#include "sha1.h"

__global__ void computeHMAC_SHA1(char *buf, char *keys,  uint32_t *offsets, uint16_t *lengths, uint32_t *outputs, int N, uint8_t *checkbits)
{
	uint32_t w_register[16];

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N) {
		uint32_t *w = w_register;
		hash_digest_t h;
		uint32_t offset = offsets[index];
		uint16_t length = lengths[index];
		uint32_t *out = outputs + 5 * index;


		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x36363636;
		xorpads(w, (uint32_t*)(keys + 64 * index));


		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		//SHA1 compute on ipad
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA1 compute on mesage
		unsigned num_iter = (length + 63 + 9) / 64;
		for (unsigned i = 0; i < num_iter; i++)
			computeSHA1Block(buf + offset , w, i * 64  , length , h);

		*(out)   = swap(h.h1);
		*(out+1) = swap(h.h2);
		*(out+2) = swap(h.h3);
		*(out+3) = swap(h.h4);
		*(out+4) = swap(h.h5);

		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x5c5c5c5c;

		xorpads(w, (uint32_t*)(keys + 64 * index));

		//SHA 1 compute on opads
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA 1 compute on (hash of ipad|m)
		computeSHA1Block((char*)out, w, 0, 20, h);

		*(out)   = swap(h.h1);
		*(out+1) = swap(h.h2);
		*(out+2) = swap(h.h3);
		*(out+3) = swap(h.h4);
		*(out+4) = swap(h.h5);
	}
        __syncthreads();

	if (threadIdx.x == 0)
		*(checkbits + blockIdx.x) = 1;

}

extern "C" void hmac_sha1_gpu(char *buf, char *keys,  uint32_t *offsets, uint16_t *lengths,
		   uint32_t *outputs, int N, uint8_t *checkbits,
		   unsigned threads_per_blk, hipStream_t stream)
{
	int num_blks = (N + threads_per_blk - 1) / threads_per_blk;
	if (stream == 0) {
		computeHMAC_SHA1<<<num_blks, threads_per_blk>>>(
		       buf, keys, offsets, lengths, outputs, N, checkbits);
	} else  {
		computeHMAC_SHA1<<<num_blks, threads_per_blk, 0, stream>>>(
		       buf, keys, offsets, lengths, outputs, N, checkbits);
	}
}

