#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <assert.h>

#include "aes_core.h"
#include "sha1.h"
#include "crypto_size.h"

/* AES counter mode + HMAC SHA-1, 
   the encryption of each block in AES counter mode is not parallelized in this implementation */
extern "C" __global__ void
aes_ctr_sha1_kernel(
			const uint8_t	*input_buf,
			uint8_t *output_buf,
			const uint32_t *pkt_offset,
			const uint16_t *length,
			const uint8_t *aes_keys,
			const uint8_t *hmac_keys,
			const unsigned int num_flows,
			uint8_t *checkbits)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int step = ceil(((float)num_flows) / (blockDim.x * gridDim.x));
	idx = idx * step;

/**************************************************************************
  AES Encryption is started first
 ***************************************************************************/
	__shared__ uint32_t shared_Te0[256];
	__shared__ uint32_t shared_Te1[256];
	__shared__ uint32_t shared_Te2[256];
	__shared__ uint32_t shared_Te3[256];
	__shared__ uint32_t shared_Rcon[10];

	/* initialize T boxes */
	for (unsigned i = 0; i * blockDim.x < 256 ; i ++) {
		unsigned index = threadIdx.x + i * blockDim.x;
		if (index >= 256)
			break;
		shared_Te0[index] = Te0_ConstMem[index];
		shared_Te1[index] = Te1_ConstMem[index];
		shared_Te2[index] = Te2_ConstMem[index];
		shared_Te3[index] = Te3_ConstMem[index];
	}

	for(unsigned i = 0; i * blockDim.x < 10; i ++){
		int index = threadIdx.x + blockDim.x * i;
		if(index < 10){
			shared_Rcon[index] = rcon[index];
		}
	}

	/* ----debug-----*/
	if (idx >= num_flows) {
		return;
	}

	/* make sure T boxes have been initialized. */
	__syncthreads();

	uint16_t len;

	for (unsigned i = 0; (i < step) && (idx < num_flows); i++, idx++) {

		/* ============================== AES CTR =============================== */
		uint64_t counter[2] = {0, 0};

		/* Locate data */
		const uint8_t *in  = pkt_offset[idx] + input_buf;
		uint8_t *out       = pkt_offset[idx] + output_buf;
		const uint8_t *key = idx * AES_KEY_SIZE + aes_keys;

		/* Encrypt using cbc mode */
		len = length[idx];

		while (len >= AES_BLOCK_SIZE) {
			/* Update counter for each block */
			counter[0] ++;
			if (counter[0] == 0) counter[1] ++;

			AES_128_encrypt((uint8_t *)counter, out, key,
					shared_Te0, shared_Te1, shared_Te2, shared_Te3, shared_Rcon);

			*((uint64_t*)out)       = *((uint64_t*)in)       ^ *((uint64_t*)out);
			*(((uint64_t*)out) + 1) = *(((uint64_t*)in) + 1) ^ *(((uint64_t*)out) + 1);

			len -= AES_BLOCK_SIZE;
			in  += AES_BLOCK_SIZE;
			out += AES_BLOCK_SIZE;
		}


		if (len) {
			counter[0] ++;
			if (counter[0] == 0) counter[1] ++;

			AES_128_encrypt((uint8_t *)counter, out, key,
					shared_Te0, shared_Te1, shared_Te2, shared_Te3, shared_Rcon);

			for(unsigned n = 0; n < len; ++n)
				out[n] = in[n] ^ out[n];
		}

		__syncthreads();

		/**************************************************************************
		  AES Encryption completed, Now we start SHA-1 Calculation
		 ***************************************************************************/
		uint32_t w_register[16];

		uint32_t *w = w_register;
		hash_digest_t h;
		uint32_t offset = pkt_offset[idx];
		len = length[idx];
		uint16_t sha1_output_pos = (len + 3) & ~0x03;
		//printf("len %d, pad_len %d, sha1_output_pos %d\n", len, pkt_offset[idx+1]-pkt_offset[idx], sha1_output_pos);
		if (sha1_output_pos > pkt_offset[idx+1] - pkt_offset[idx] - HMAC_TAG_SIZE) {
			printf("ERROR: Sha1 output position exceeds the packet boundary! len %d, output %d, pad_len %d, hmac %d\n",
					len, sha1_output_pos, pkt_offset[idx+1] - pkt_offset[idx], HMAC_TAG_SIZE);
			return;
		}
		uint32_t *sha1_out = (uint32_t *)(input_buf + offset + sha1_output_pos);

		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x36363636;
		xorpads(w, (uint32_t *)(hmac_keys + HMAC_KEY_SIZE * idx));

		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		//SHA1 compute on ipad
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA1 compute on message
		unsigned num_iter = (pkt_offset[idx+1] - pkt_offset[idx]) / 64;
		for (unsigned i = 0; i < num_iter; i ++)
			computeSHA1Block((char *)(input_buf + offset), w, i * 64, len, h);

		/* In SRTP, sha1_out has only 80 bits output 32+32+16 = 80 */
		*(sha1_out)   = swap(h.h1);
		*(sha1_out+1) = swap(h.h2);
		*(sha1_out+2) = swap(h.h3);
		*(sha1_out+3) = swap(h.h4);
		*(sha1_out+4) = swap(h.h5);

		h.h1 = 0x67452301;
		h.h2 = 0xEFCDAB89;
		h.h3 = 0x98BADCFE;
		h.h4 = 0x10325476;
		h.h5 = 0xC3D2E1F0;

		for (unsigned i = 0; i < 16; i++)
			w[i] = 0x5c5c5c5c;

		xorpads(w, (uint32_t*)(hmac_keys + HMAC_KEY_SIZE * idx));

		//SHA 1 compute on opads
		computeSHA1Block((char*)w, w, 0, 64, h);

		//SHA 1 compute on (hash of ipad|m)
		computeSHA1Block((char*)sha1_out, w, 0, 20, h);

		*(sha1_out)   = swap(h.h1);
		*(sha1_out+1) = swap(h.h2);
		*(sha1_out+2) = swap(h.h3);
		*(sha1_out+3) = swap(h.h4);
		*(sha1_out+4) = swap(h.h5);

		__syncthreads();
	}
	return;
}

extern "C" void co_aes_sha1_gpu(
			const uint8_t		*in,
			uint8_t				*out,
			const uint32_t		*pkt_offset,
			const uint16_t		*actual_length,
			const uint8_t		*aes_keys,
			const uint8_t		*hmac_keys,
			const unsigned int 	num_flows,
			uint8_t				*checkbits,
			const unsigned int	threads_per_blk,
			const unsigned int	num_blks,
			hipStream_t stream)
{
	//printf("stream=%d, threads_per_blk =%d, num_blks = %d\n", stream, threads_per_blk, num_blks);
	if (stream == 0) {
		aes_ctr_sha1_kernel<<<num_blks, threads_per_blk>>>(
		       in, out, pkt_offset, actual_length, aes_keys, hmac_keys, num_flows, checkbits);
	} else  {
		aes_ctr_sha1_kernel<<<num_blks, threads_per_blk, 0, stream>>>(
		       in, out, pkt_offset, actual_length, aes_keys, hmac_keys, num_flows, checkbits);
	}
}

